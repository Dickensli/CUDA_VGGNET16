/*
 * conv_unnitest.cu
 * Copyright (C) 2018-06-09 Hanxiao <hah114@ucsd.edu>
 *
 * Distributed under terms of the MIT license.
 */

#include <algorithm>
#include <cctype>
#include <cmath>
#include <cstring>
#include <cstdlib>
#include <stdlib.h>
#include <fstream>
#include <iostream>
#include <stdexcept>
#include <string>
#include <vector>
#include <cstdio>
#include <hip/hip_runtime_api.h>
#include "conv.h"
#include "pool_fn.h"

#define IMAGE_WIDTH 224
#define CHANNELS 3
#define THREADBLOCK 24
#define LAYER1_PARAMS 1728 //params: (3*3*3)*64
#define LAYER1_BIAS_PARAMS 64 //bias: 64
#define LAYER2_PARAMS 36864 //params: (3*3*64)*64
#define LAYER2_BIAS_PARAMS 64 //bias: 64
#define LAYER3_PARAMS 73728 //params: (3*3*64)*128
#define LAYER3_BIAS_PARAMS 128 //bias: 128
#define LAYER4_PARAMS 147456 //params: (3*3*128)*128
#define LAYER4_BIAS_PARAMS 128 //bias: 128
#define LAYER5_PARAMS 294912 //params: (3*3*128)*256
#define LAYER5_BIAS_PARAMS 256 //bias: 256
#define LAYER6_PARAMS 589824 //params: (3*3*256)*256
#define LAYER6_BIAS_PARAMS 256 //bias: 256
#define LAYER7_PARAMS 589824 //params: (3*3*256)*256
#define LAYER7_BIAS_PARAMS 256 //bias: 256
#define LAYER8_PARAMS 1179648 //params: (3*3*256)*512
#define LAYER8_BIAS_PARAMS 512 //bias: 512
#define LAYER9_PARAMS 2359296 //params: (3*3*512)*512
#define LAYER9_BIAS_PARAMS 512 //bias: 512
#define LAYER10_PARAMS 2359296 //params: (3*3*512)*512
#define LAYER10_BIAS_PARAMS 512 //bias: 512
#define LAYER11_PARAMS 2359296 //params: (3*3*512)*512
#define LAYER11_BIAS_PARAMS 512 //bias: 512
#define LAYER12_PARAMS 2359296 //params: (3*3*512)*512
#define LAYER12_BIAS_PARAMS 512 //bias: 512
#define LAYER13_PARAMS 2359296 //params: (3*3*512)*512
#define LAYER13_BIAS_PARAMS 512 //bias: 512
#define LAYER14_PARAMS 102760448 //params: (7*7*512)*4096
#define LAYER14_BIAS_PARAMS 4096
#define LAYER15_PARAMS 16777216 //params: 4096*4096
#define LAYER15_BIAS_PARAMS 4096
#define LAYER16_PARAMS 4096000 //params: 4096*4096
#define LAYER16_BIAS_PARAMS 1000
#define MASK_WIDTH 3
#include "utils.h"

using namespace std;

void ConvertInput(float *&Data_Layer_CPU);
void LoadImageNetClass(char **image_class, char *file_path);
void LoadInput(float *Data_Layer_CPU,char* file_path);
void InitWeights_Biases(float *Weights_CPU, int size, char* file_path);


int main(int argc, char **argv)
{
    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));

    if(argc != 2){
        cout << "Usage ./vgg_net file_path" << endl;
        return -1;
    }
    char * file_path = argv[1];

    int x = 0;
    // timing setup
    hipEvent_t start_event, stop_event;
    hipEventCreate(&start_event) ;
    hipEventCreate(&stop_event);
    float milliseconds = 0;
    float totalTime = 0;

    //Allocation of host memory for weights
    float *Layer1_Weights_CPU = (float*) malloc (LAYER1_PARAMS * sizeof(float)); //no. of features in nth layer
    float *Layer1_Weights_Bias_CPU = (float*) malloc (LAYER1_BIAS_PARAMS * sizeof(float));
    float *Layer2_Weights_CPU = (float*) malloc (LAYER2_PARAMS * sizeof(float)); // no. of features in nth layer
    float *Layer2_Weights_Bias_CPU = (float*) malloc (LAYER2_BIAS_PARAMS * sizeof(float));
    float *Layer3_Weights_CPU = (float*) malloc (LAYER3_PARAMS * sizeof(float));
    float *Layer3_Weights_Bias_CPU = (float*) malloc (LAYER3_BIAS_PARAMS * sizeof(float));
    float *Layer4_Weights_CPU = (float*) malloc (LAYER4_PARAMS * sizeof(float));
    float *Layer4_Weights_Bias_CPU = (float*) malloc (LAYER4_BIAS_PARAMS * sizeof(float));
    float *Layer5_Weights_CPU = (float*) malloc (LAYER5_PARAMS * sizeof(float));
    float *Layer5_Weights_Bias_CPU = (float*) malloc (LAYER5_BIAS_PARAMS * sizeof(float));
    float *Layer6_Weights_CPU = (float*) malloc (LAYER6_PARAMS * sizeof(float));
    float *Layer6_Weights_Bias_CPU = (float*) malloc (LAYER6_BIAS_PARAMS * sizeof(float));
    float *Layer7_Weights_CPU = (float*) malloc (LAYER7_PARAMS * sizeof(float));
    float *Layer7_Weights_Bias_CPU = (float*) malloc (LAYER7_BIAS_PARAMS * sizeof(float));
    float *Layer8_Weights_CPU = (float*) malloc (LAYER8_PARAMS * sizeof(float));
    float *Layer8_Weights_Bias_CPU = (float*) malloc (LAYER8_BIAS_PARAMS * sizeof(float));
    float *Layer9_Weights_CPU = (float*) malloc (LAYER9_PARAMS * sizeof(float));
    float *Layer9_Weights_Bias_CPU = (float*) malloc (LAYER9_BIAS_PARAMS * sizeof(float));
    float *Layer10_Weights_CPU = (float*) malloc (LAYER10_PARAMS * sizeof(float));
    float *Layer10_Weights_Bias_CPU = (float*) malloc (LAYER10_BIAS_PARAMS * sizeof(float));
    float *Layer11_Weights_CPU = (float*) malloc (LAYER11_PARAMS * sizeof(float));
    float *Layer11_Weights_Bias_CPU = (float*) malloc (LAYER11_BIAS_PARAMS * sizeof(float));
    float *Layer12_Weights_CPU = (float*) malloc (LAYER12_PARAMS * sizeof(float));
    float *Layer12_Weights_Bias_CPU = (float*) malloc (LAYER12_BIAS_PARAMS * sizeof(float));
    float *Layer13_Weights_CPU = (float*) malloc (LAYER13_PARAMS * sizeof(float));
    float *Layer13_Weights_Bias_CPU = (float*) malloc (LAYER13_BIAS_PARAMS * sizeof(float));
    float *Layer14_Weights_CPU = (float*) malloc (LAYER14_PARAMS * sizeof(float));
    float *Layer14_Weights_Bias_CPU = (float*) malloc (LAYER14_BIAS_PARAMS * sizeof(float));
    float *Layer15_Weights_CPU = (float*) malloc (LAYER15_PARAMS * sizeof(float));
    float *Layer15_Weights_Bias_CPU = (float*) malloc (LAYER15_BIAS_PARAMS * sizeof(float));
    float *Layer16_Weights_CPU = (float*) malloc (LAYER16_PARAMS * sizeof(float));
    float *Layer16_Weights_Bias_CPU = (float*) malloc (LAYER16_BIAS_PARAMS * sizeof(float));

    float *Data_Layer_CPU = (float*) malloc (CHANNELS*IMAGE_WIDTH*IMAGE_WIDTH * sizeof(float));

    InitWeights_Biases(Layer1_Weights_CPU,LAYER1_PARAMS, (char *)"data/conv1_1_v.txt");
    InitWeights_Biases(Layer1_Weights_Bias_CPU,LAYER1_BIAS_PARAMS, (char *)"data/conv1_1_v_bias.txt");
    InitWeights_Biases(Layer2_Weights_CPU,LAYER2_PARAMS, (char *)"data/conv1_2_v.txt");
    InitWeights_Biases(Layer2_Weights_Bias_CPU,LAYER2_BIAS_PARAMS, (char *)"data/conv1_2_v_bias.txt");
    InitWeights_Biases(Layer3_Weights_CPU,LAYER3_PARAMS, (char *)"data/conv2_1_v.txt");
    InitWeights_Biases(Layer3_Weights_Bias_CPU,LAYER3_BIAS_PARAMS, (char *)"data/conv2_1_v_bias.txt");
    InitWeights_Biases(Layer4_Weights_CPU,LAYER4_PARAMS, (char *)"data/conv2_2_v.txt");
    InitWeights_Biases(Layer4_Weights_Bias_CPU,LAYER4_BIAS_PARAMS, (char *)"data/conv2_2_v_bias.txt");
    InitWeights_Biases(Layer5_Weights_CPU,LAYER5_PARAMS, (char *)"data/conv3_1_v.txt");
    InitWeights_Biases(Layer5_Weights_Bias_CPU,LAYER5_BIAS_PARAMS, (char *)"data/conv3_1_v_bias.txt");
    InitWeights_Biases(Layer6_Weights_CPU,LAYER6_PARAMS, (char *)"data/conv3_2_v.txt");
    InitWeights_Biases(Layer6_Weights_Bias_CPU,LAYER6_BIAS_PARAMS, (char *)"data/conv3_2_v_bias.txt");
    InitWeights_Biases(Layer7_Weights_CPU,LAYER7_PARAMS, (char *)"data/conv3_3_v.txt");
    InitWeights_Biases(Layer7_Weights_Bias_CPU,LAYER7_BIAS_PARAMS, (char *)"data/conv3_3_v_bias.txt");
    InitWeights_Biases(Layer8_Weights_CPU,LAYER8_PARAMS, (char *)"data/conv4_1_v.txt");
    InitWeights_Biases(Layer8_Weights_Bias_CPU,LAYER8_BIAS_PARAMS, (char *)"data/conv4_1_v_bias.txt");
    InitWeights_Biases(Layer9_Weights_CPU,LAYER9_PARAMS, (char *)"data/conv4_2_v.txt");
    InitWeights_Biases(Layer9_Weights_Bias_CPU,LAYER9_BIAS_PARAMS, (char *)"data/conv4_2_v_bias.txt");
    InitWeights_Biases(Layer10_Weights_CPU,LAYER10_PARAMS, (char *)"data/conv4_3_v.txt");
    InitWeights_Biases(Layer10_Weights_Bias_CPU,LAYER10_BIAS_PARAMS, (char *)"data/conv4_3_v_bias.txt");
    InitWeights_Biases(Layer11_Weights_CPU,LAYER11_PARAMS, (char *)"data/conv5_1_v.txt");
    InitWeights_Biases(Layer11_Weights_Bias_CPU,LAYER11_BIAS_PARAMS, (char *)"data/conv5_1_v_bias.txt");
    InitWeights_Biases(Layer12_Weights_CPU,LAYER12_PARAMS, (char *)"data/conv5_2_v.txt");
    InitWeights_Biases(Layer12_Weights_Bias_CPU,LAYER12_BIAS_PARAMS, (char *)"data/conv5_2_v_bias.txt");
    InitWeights_Biases(Layer13_Weights_CPU,LAYER13_PARAMS, (char *)"data/conv5_3_v.txt");
    InitWeights_Biases(Layer13_Weights_Bias_CPU,LAYER13_BIAS_PARAMS, (char *)"data/conv5_3_v_bias.txt");
    InitWeights_Biases(Layer14_Weights_CPU,LAYER14_PARAMS, (char *)"data/fc6_v.txt");
    InitWeights_Biases(Layer14_Weights_Bias_CPU,LAYER14_BIAS_PARAMS, (char *)"data/fc6_v_bias.txt");
    InitWeights_Biases(Layer15_Weights_CPU,LAYER15_PARAMS, (char *)"data/fc7_v.txt");
    InitWeights_Biases(Layer15_Weights_Bias_CPU,LAYER15_BIAS_PARAMS, (char *)"data/fc7_v_bias.txt");
    InitWeights_Biases(Layer16_Weights_CPU,LAYER16_PARAMS, (char *)"data/fc8_v.txt");
    InitWeights_Biases(Layer16_Weights_Bias_CPU,LAYER16_BIAS_PARAMS, (char *)"data/fc8_v_bias.txt");

    LoadInput(Data_Layer_CPU,file_path);
    ConvertInput(Data_Layer_CPU);

    //Allocate device GMEM input
    float *Data_Layer_GPU;
    hipMalloc(&Data_Layer_GPU, CHANNELS*IMAGE_WIDTH*IMAGE_WIDTH * sizeof(float));
    hipMemcpy(Data_Layer_GPU, Data_Layer_CPU, CHANNELS*IMAGE_WIDTH*IMAGE_WIDTH * sizeof(float), hipMemcpyHostToDevice);

    int inDimention, outDimention, size;
    //layer 1
    inDimention = 3; outDimention = 64; size = 224;
    float *Conv_Layer1_Features;
    float *Layer1_Weights_GPU, *Layer1_Weights_Bias_GPU;
    hipMalloc(&Layer1_Weights_GPU, LAYER1_PARAMS * sizeof(float));
    hipMalloc(&Layer1_Weights_Bias_GPU, LAYER1_BIAS_PARAMS * sizeof(float));
    hipMalloc(&Conv_Layer1_Features, size * size * outDimention * sizeof(float));
    hipMemcpy(Layer1_Weights_GPU, Layer1_Weights_CPU, LAYER1_PARAMS * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(Layer1_Weights_Bias_GPU, Layer1_Weights_Bias_CPU, LAYER1_BIAS_PARAMS * sizeof(float), hipMemcpyHostToDevice);

    hipEventRecord(start_event, 0);
    ConvLayer4<<<THREADBLOCK, dim3(32, 32, 1), (3 * 3 * inDimention) * sizeof(float)>>>(Data_Layer_GPU, Conv_Layer1_Features, Layer1_Weights_GPU, Layer1_Weights_Bias_GPU, inDimention, size, outDimention);
    CHECK(hipDeviceSynchronize());
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&milliseconds, start_event, stop_event);
    cout << "Conv1: " << milliseconds / 1000 << endl;
    totalTime += milliseconds / 1000;
    hipFree(Data_Layer_GPU);

    //layer 2
    inDimention = outDimention; outDimention = 64; size = 224;
    float *Conv_Layer2_Features;
    float *Layer2_Weights_GPU, *Layer2_Weights_Bias_GPU;
    hipMalloc(&Layer2_Weights_GPU, LAYER2_PARAMS * sizeof(float));
    hipMalloc(&Layer2_Weights_Bias_GPU, LAYER2_BIAS_PARAMS * sizeof(float));
    hipMalloc(&Conv_Layer2_Features, size * size * outDimention * sizeof(float));
    hipMemcpy(Layer2_Weights_GPU, Layer2_Weights_CPU, LAYER2_PARAMS * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(Layer2_Weights_Bias_GPU, Layer2_Weights_Bias_CPU, LAYER2_BIAS_PARAMS * sizeof(float), hipMemcpyHostToDevice);

    hipEventRecord(start_event, 0);
    //ConvLayer4<<<THREADBLOCK, dim3(32, 32, 1), (3 * 3 * inDimention) * sizeof(float)>>>(Conv_Layer1_Features, Conv_Layer2_Features, Layer2_Weights_GPU, Layer2_Weights_Bias_GPU, inDimention, size, outDimention);
    //ConvLayer2<<<outDimention, dim3(32, 32, 1)>>>(Conv_Layer1_Features, Conv_Layer2_Features, Layer2_Weights_GPU, Layer2_Weights_Bias_GPU, inDimention, size, outDimention);
    ConvLayer1<<<dim3(7,7,1), dim3(32, 32, 1)>>>(Conv_Layer1_Features, Conv_Layer2_Features, Layer2_Weights_GPU, Layer2_Weights_Bias_GPU, inDimention, size, outDimention);
    CHECK(hipDeviceSynchronize());
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&milliseconds, start_event, stop_event);
    cout << "Conv2: " << milliseconds / 1000 << endl;
    totalTime += milliseconds / 1000;
    hipFree(Conv_Layer1_Features);

    //pool layer 2
    float * Pool_Layer2_Features ;
    hipMalloc(&Pool_Layer2_Features, 112 * 112 * 128 * sizeof(float));
    hipEventRecord(start_event, 0);
    pool_2<<<dim3(14, 14, 1), dim3(8, 8, 1)>>>(Conv_Layer2_Features, Pool_Layer2_Features, 224, 224 * 64);
    CHECK(hipDeviceSynchronize());
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&milliseconds, start_event, stop_event);
    cout << "Pool2: " << milliseconds / 1000 << endl;
    totalTime += milliseconds / 1000;
    hipFree(Conv_Layer2_Features);

    //layer 3
    inDimention = outDimention; outDimention = 128; size = 112;
    float *Conv_Layer3_Features;
    float *Layer3_Weights_GPU, *Layer3_Weights_Bias_GPU;
    hipMalloc(&Layer3_Weights_GPU, LAYER3_PARAMS * sizeof(float));
    hipMalloc(&Layer3_Weights_Bias_GPU, LAYER3_BIAS_PARAMS * sizeof(float));
    hipMalloc(&Conv_Layer3_Features, size * size * outDimention * sizeof(float));
    hipMemcpy(Layer3_Weights_GPU, Layer3_Weights_CPU, LAYER3_PARAMS * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(Layer3_Weights_Bias_GPU, Layer3_Weights_Bias_CPU, LAYER3_BIAS_PARAMS * sizeof(float), hipMemcpyHostToDevice);

    hipEventRecord(start_event, 0);
    //ConvLayer4<<<THREADBLOCK, dim3(32, 32, 1), (3 * 3 * inDimention) * sizeof(float)>>>(Pool_Layer2_Features, Conv_Layer3_Features, Layer3_Weights_GPU, Layer3_Weights_Bias_GPU, inDimention, size, outDimention);
    ConvLayer2<<<outDimention, dim3(32, 32, 1)>>>(Pool_Layer2_Features, Conv_Layer3_Features, Layer3_Weights_GPU, Layer3_Weights_Bias_GPU, inDimention, size, outDimention);
    CHECK(hipDeviceSynchronize());
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&milliseconds, start_event, stop_event);
    cout << "Conv3: " << milliseconds / 1000 << endl;
    totalTime += milliseconds / 1000;
    hipFree(Pool_Layer2_Features);

    //layer 4
    inDimention = outDimention; outDimention = 128; size = 112;
    float *Conv_Layer4_Features;
    float *Layer4_Weights_GPU, *Layer4_Weights_Bias_GPU;
    hipMalloc(&Layer4_Weights_GPU, LAYER4_PARAMS * sizeof(float));
    hipMalloc(&Layer4_Weights_Bias_GPU, LAYER4_BIAS_PARAMS * sizeof(float));
    hipMalloc(&Conv_Layer4_Features, size * size * outDimention * sizeof(float));
    hipMemcpy(Layer4_Weights_GPU, Layer4_Weights_CPU, LAYER4_PARAMS * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(Layer4_Weights_Bias_GPU, Layer4_Weights_Bias_CPU, LAYER4_BIAS_PARAMS * sizeof(float), hipMemcpyHostToDevice);

    hipEventRecord(start_event, 0);
    //ConvLayer4<<<THREADBLOCK, dim3(32, 32, 1), (3 * 3 * inDimention) * sizeof(float)>>>(Conv_Layer3_Features, Conv_Layer4_Features, Layer4_Weights_GPU, Layer4_Weights_Bias_GPU, inDimention, size, outDimention);
    ConvLayer2<<<outDimention, dim3(32, 32, 1)>>>(Conv_Layer3_Features, Conv_Layer4_Features, Layer4_Weights_GPU, Layer4_Weights_Bias_GPU, inDimention, size, outDimention);
    CHECK(hipDeviceSynchronize());
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&milliseconds, start_event, stop_event);
    cout << "Conv4: " << milliseconds / 1000 << endl;
    totalTime += milliseconds / 1000;
    hipFree(Conv_Layer3_Features);


    //pool layer 4
    float * Pool_Layer4_Features ;
    hipMalloc(&Pool_Layer4_Features, 56 * 56 * 256 * sizeof(float));
    hipEventRecord(start_event, 0);
    pool_2<<<dim3(7, 7, 1), dim3(8, 8, 1)>>>(Conv_Layer4_Features, Pool_Layer4_Features, 112, 112 * 128);
    CHECK(hipDeviceSynchronize());
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&milliseconds, start_event, stop_event);
    cout << "Pool4: " << milliseconds / 1000 << endl;
    totalTime += milliseconds / 1000;
    hipFree(Conv_Layer4_Features);

    //layer 5 
    inDimention = outDimention; outDimention = 256; size = 56;
    float *Conv_Layer5_Features;
    float *Layer5_Weights_GPU, *Layer5_Weights_Bias_GPU;
    hipMalloc(&Layer5_Weights_GPU, LAYER5_PARAMS * sizeof(float));
    hipMalloc(&Layer5_Weights_Bias_GPU, LAYER5_BIAS_PARAMS * sizeof(float));
    hipMalloc(&Conv_Layer5_Features, size * size * outDimention * sizeof(float));
    hipMemcpy(Layer5_Weights_GPU, Layer5_Weights_CPU, LAYER5_PARAMS * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(Layer5_Weights_Bias_GPU, Layer5_Weights_Bias_CPU, LAYER5_BIAS_PARAMS * sizeof(float), hipMemcpyHostToDevice);

    hipEventRecord(start_event, 0);
    ConvLayer4<<<THREADBLOCK, dim3(32, 32, 1), (3 * 3 * inDimention) * sizeof(float)>>>(Pool_Layer4_Features, Conv_Layer5_Features, Layer5_Weights_GPU, Layer5_Weights_Bias_GPU, inDimention, size, outDimention);
    CHECK(hipDeviceSynchronize());
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&milliseconds, start_event, stop_event);
    cout << "Conv5: " << milliseconds / 1000 << endl;
    totalTime += milliseconds / 1000;
    hipFree(Pool_Layer4_Features);

    //layer 6
    inDimention = outDimention; outDimention = 256; size = 56;
    float *Conv_Layer6_Features;
    float *Layer6_Weights_GPU, *Layer6_Weights_Bias_GPU;
    hipMalloc(&Layer6_Weights_GPU, LAYER6_PARAMS * sizeof(float));
    hipMalloc(&Layer6_Weights_Bias_GPU, LAYER6_BIAS_PARAMS * sizeof(float));
    hipMalloc(&Conv_Layer6_Features, size * size * outDimention * sizeof(float));
    hipMemcpy(Layer6_Weights_GPU, Layer6_Weights_CPU, LAYER6_PARAMS * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(Layer6_Weights_Bias_GPU, Layer6_Weights_Bias_CPU, LAYER6_BIAS_PARAMS * sizeof(float), hipMemcpyHostToDevice);

    hipEventRecord(start_event, 0);
    ConvLayer4<<<THREADBLOCK, dim3(32, 32, 1), (3 * 3 * inDimention) * sizeof(float)>>>(Conv_Layer5_Features, Conv_Layer6_Features, Layer6_Weights_GPU, Layer6_Weights_Bias_GPU, inDimention, size, outDimention);
    CHECK(hipDeviceSynchronize());
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&milliseconds, start_event, stop_event);
    cout << "Conv6: " << milliseconds / 1000 << endl;
    totalTime += milliseconds / 1000;
    hipFree(Conv_Layer5_Features);

    //layer 7
    inDimention = outDimention; outDimention = 256; size = 56;
    float *Conv_Layer7_Features;
    float *Layer7_Weights_GPU, *Layer7_Weights_Bias_GPU;
    hipMalloc(&Layer7_Weights_GPU, LAYER7_PARAMS * sizeof(float));
    hipMalloc(&Layer7_Weights_Bias_GPU, LAYER7_BIAS_PARAMS * sizeof(float));
    hipMalloc(&Conv_Layer7_Features, size * size * outDimention * sizeof(float));
    hipMemcpy(Layer7_Weights_GPU, Layer7_Weights_CPU, LAYER7_PARAMS * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(Layer7_Weights_Bias_GPU, Layer7_Weights_Bias_CPU, LAYER7_BIAS_PARAMS * sizeof(float), hipMemcpyHostToDevice);

    hipEventRecord(start_event, 0);
    ConvLayer4<<<THREADBLOCK, dim3(32, 32, 1), (3 * 3 * inDimention) * sizeof(float)>>>(Conv_Layer6_Features, Conv_Layer7_Features, Layer7_Weights_GPU, Layer7_Weights_Bias_GPU, inDimention, size, outDimention);
    CHECK(hipDeviceSynchronize());
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&milliseconds, start_event, stop_event);
    cout << "Conv7: " << milliseconds / 1000 << endl;
    totalTime += milliseconds / 1000;
    hipFree(Conv_Layer6_Features);

    //pool layer 7
    float * Pool_Layer7_Features ;
    hipMalloc(&Pool_Layer7_Features, 28 * 28 * 512 * sizeof(float));
    hipEventRecord(start_event, 0);
    pool_2<<<dim3(4, 4, 1), dim3(8, 8, 1)>>>(Conv_Layer7_Features, Pool_Layer7_Features, 56, 56 * 256);
    CHECK(hipDeviceSynchronize());
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&milliseconds, start_event, stop_event);
    cout << "Pool7: " << milliseconds / 1000 << endl;
    totalTime += milliseconds / 1000;
    hipFree(Conv_Layer7_Features);

    //layer 8
    inDimention = outDimention; outDimention = 512; size = 28;
    float *Conv_Layer8_Features;
    float *Layer8_Weights_GPU, *Layer8_Weights_Bias_GPU;
    hipMalloc(&Layer8_Weights_GPU, LAYER8_PARAMS * sizeof(float));
    hipMalloc(&Layer8_Weights_Bias_GPU, LAYER8_BIAS_PARAMS * sizeof(float));
    hipMalloc(&Conv_Layer8_Features, size * size * outDimention * sizeof(float));
    hipMemcpy(Layer8_Weights_GPU, Layer8_Weights_CPU, LAYER8_PARAMS * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(Layer8_Weights_Bias_GPU, Layer8_Weights_Bias_CPU, LAYER8_BIAS_PARAMS * sizeof(float), hipMemcpyHostToDevice);

    hipEventRecord(start_event, 0);
    ConvLayer4<<<THREADBLOCK, dim3(32, 32, 1), (3 * 3 * inDimention) * sizeof(float)>>>(Pool_Layer7_Features, Conv_Layer8_Features, Layer8_Weights_GPU, Layer8_Weights_Bias_GPU, inDimention, size, outDimention);
    CHECK(hipDeviceSynchronize());
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&milliseconds, start_event, stop_event);
    cout << "Conv8: " << milliseconds / 1000 << endl;
    totalTime += milliseconds / 1000;
    hipFree(Pool_Layer7_Features);

    //layer 9
    inDimention = outDimention; outDimention = 512; size = 28;
    float * Conv_Layer9_Features;
    float *Layer9_Weights_GPU, *Layer9_Weights_Bias_GPU;
    hipMalloc(&Layer9_Weights_GPU, LAYER9_PARAMS * sizeof(float));
    hipMalloc(&Layer9_Weights_Bias_GPU, LAYER9_BIAS_PARAMS * sizeof(float));
    hipMalloc(&Conv_Layer9_Features, size * size * outDimention * sizeof(float));
    hipMemcpy(Layer9_Weights_GPU, Layer9_Weights_CPU, LAYER9_PARAMS * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(Layer9_Weights_Bias_GPU, Layer9_Weights_Bias_CPU, LAYER9_BIAS_PARAMS * sizeof(float), hipMemcpyHostToDevice);

    hipEventRecord(start_event, 0);
    ConvLayer4<<<THREADBLOCK, dim3(32, 32, 1), (3 * 3 * inDimention) * sizeof(float)>>>(Conv_Layer8_Features, Conv_Layer9_Features, Layer9_Weights_GPU, Layer9_Weights_Bias_GPU, inDimention, size, outDimention);
    //ConvLayer3_1<<<THREADBLOCK, inDimention, (3 * 3 * inDimention + inDimention / 32) * sizeof(float)>>>(Conv_Layer8_Features, Conv_Layer9_Features, Layer9_Weights_GPU, Layer9_Weights_Bias_GPU, inDimention, size, outDimention);
    CHECK(hipDeviceSynchronize());
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&milliseconds, start_event, stop_event);
    cout << "Conv9: " << milliseconds / 1000 << endl;
    totalTime += milliseconds / 1000;
    hipFree(Conv_Layer8_Features);

    //layer 10
    inDimention = outDimention; outDimention = 512; size = 28;
    float *Conv_Layer10_Features;
    float *Layer10_Weights_GPU, *Layer10_Weights_Bias_GPU;
    hipMalloc(&Layer10_Weights_GPU, LAYER10_PARAMS * sizeof(float));
    hipMalloc(&Layer10_Weights_Bias_GPU, LAYER10_BIAS_PARAMS * sizeof(float));
    hipMalloc(&Conv_Layer10_Features, size * size * outDimention * sizeof(float));
    hipMemcpy(Layer10_Weights_GPU, Layer10_Weights_CPU, LAYER10_PARAMS * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(Layer10_Weights_Bias_GPU, Layer10_Weights_Bias_CPU, LAYER10_BIAS_PARAMS * sizeof(float), hipMemcpyHostToDevice);

    hipEventRecord(start_event, 0);
    ConvLayer4<<<THREADBLOCK, dim3(32, 32, 1), (3 * 3 * inDimention) * sizeof(float)>>>(Conv_Layer9_Features, Conv_Layer10_Features, Layer10_Weights_GPU, Layer10_Weights_Bias_GPU, inDimention, size, outDimention);
    //ConvLayer3_1<<<THREADBLOCK, inDimention, (3 * 3 * inDimention + inDimention / 32) * sizeof(float)>>>(Conv_Layer9_Features, Conv_Layer10_Features, Layer10_Weights_GPU, Layer10_Weights_Bias_GPU, inDimention, size, outDimention);
    CHECK(hipDeviceSynchronize());
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&milliseconds, start_event, stop_event);
    cout << "Conv10: " << milliseconds / 1000 << endl;
    totalTime += milliseconds / 1000;
    hipFree(Conv_Layer9_Features);

    //pool layer 10
    float * Pool_Layer10_Features ;
    hipMalloc(&Pool_Layer10_Features, 14 * 14 * 512 * sizeof(float));
    hipEventRecord(start_event, 0);
    pool_2<<<dim3(2, 2, 1), dim3(8, 8, 1)>>>(Conv_Layer10_Features, Pool_Layer10_Features, 28, 28 * 512);
    CHECK(hipDeviceSynchronize());
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&milliseconds, start_event, stop_event);
    cout << "Pool10: " << milliseconds / 1000 << endl;
    totalTime += milliseconds / 1000;
    hipFree(Conv_Layer10_Features);

    //layer 11
    inDimention = outDimention; outDimention = 512; size = 14;
    float *Conv_Layer11_Features;
    float *Layer11_Weights_GPU, *Layer11_Weights_Bias_GPU;
    hipMalloc(&Layer11_Weights_GPU, LAYER11_PARAMS * sizeof(float));
    hipMalloc(&Layer11_Weights_Bias_GPU, LAYER11_BIAS_PARAMS * sizeof(float));
    hipMalloc(&Conv_Layer11_Features, size * size * outDimention * sizeof(float));
    hipMemcpy(Layer11_Weights_GPU, Layer11_Weights_CPU, LAYER11_PARAMS * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(Layer11_Weights_Bias_GPU, Layer11_Weights_Bias_CPU, LAYER11_BIAS_PARAMS * sizeof(float), hipMemcpyHostToDevice);

    hipEventRecord(start_event, 0);
    //ConvLayer4<<<THREADBLOCK, dim3(32, 32, 1), (3 * 3 * inDimention) * sizeof(float)>>>(Pool_Layer10_Features, Conv_Layer11_Features, Layer11_Weights_GPU, Layer11_Weights_Bias_GPU, inDimention, size, outDimention);
    ConvLayer3_1<<<THREADBLOCK, inDimention, (3 * 3 * inDimention + inDimention / 32) * sizeof(float)>>>(Pool_Layer10_Features, Conv_Layer11_Features, Layer11_Weights_GPU, Layer11_Weights_Bias_GPU, inDimention, size, outDimention);
    CHECK(hipDeviceSynchronize());
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&milliseconds, start_event, stop_event);
    cout << "Conv11: " << milliseconds / 1000 << endl;
    totalTime += milliseconds / 1000;
    hipFree(Pool_Layer10_Features);

    //layer 12
    inDimention = outDimention; outDimention = 512; size = 14;
    float *Conv_Layer12_Features;
    float *Layer12_Weights_GPU, *Layer12_Weights_Bias_GPU;
    hipMalloc(&Layer12_Weights_GPU, LAYER12_PARAMS * sizeof(float));
    hipMalloc(&Layer12_Weights_Bias_GPU, LAYER12_BIAS_PARAMS * sizeof(float));
    hipMalloc(&Conv_Layer12_Features, size * size * outDimention * sizeof(float));
    hipMemcpy(Layer12_Weights_GPU, Layer12_Weights_CPU, LAYER12_PARAMS * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(Layer12_Weights_Bias_GPU, Layer12_Weights_Bias_CPU, LAYER12_BIAS_PARAMS * sizeof(float), hipMemcpyHostToDevice);

    hipEventRecord(start_event, 0);
    //ConvLayer4<<<THREADBLOCK, dim3(32, 32, 1), (3 * 3 * inDimention) * sizeof(float)>>>(Conv_Layer11_Features, Conv_Layer12_Features, Layer12_Weights_GPU, Layer12_Weights_Bias_GPU, inDimention, size, outDimention);
    ConvLayer3_1<<<THREADBLOCK, inDimention, (3 * 3 * inDimention + inDimention / 32) * sizeof(float)>>>(Conv_Layer11_Features, Conv_Layer12_Features, Layer12_Weights_GPU, Layer12_Weights_Bias_GPU, inDimention, size, outDimention);
    CHECK(hipDeviceSynchronize());
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&milliseconds, start_event, stop_event);
    cout << "Conv12: " << milliseconds / 1000 << endl;
    totalTime += milliseconds / 1000;
    hipFree(Conv_Layer11_Features);

    //layer 13
    inDimention = outDimention; outDimention = 512; size = 14;
    float *Conv_Layer13_Features;
    float *Layer13_Weights_GPU, *Layer13_Weights_Bias_GPU;
    hipMalloc(&Layer13_Weights_GPU, LAYER13_PARAMS * sizeof(float));
    hipMalloc(&Layer13_Weights_Bias_GPU, LAYER13_BIAS_PARAMS * sizeof(float));
    hipMalloc(&Conv_Layer13_Features, size * size * outDimention * sizeof(float));
    hipMemcpy(Layer13_Weights_GPU, Layer13_Weights_CPU, LAYER13_PARAMS * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(Layer13_Weights_Bias_GPU, Layer13_Weights_Bias_CPU, LAYER13_BIAS_PARAMS * sizeof(float), hipMemcpyHostToDevice);

    hipEventRecord(start_event, 0);
    //ConvLayer4<<<THREADBLOCK, dim3(32, 32, 1), (3 * 3 * inDimention) * sizeof(float)>>>(Conv_Layer12_Features, Conv_Layer13_Features, Layer13_Weights_GPU, Layer13_Weights_Bias_GPU, inDimention, size, outDimention);
    ConvLayer3_1<<<THREADBLOCK, inDimention, (3 * 3 * inDimention + inDimention / 32) * sizeof(float)>>>(Conv_Layer12_Features, Conv_Layer13_Features, Layer13_Weights_GPU, Layer13_Weights_Bias_GPU, inDimention, size, outDimention);
    CHECK(hipDeviceSynchronize());
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&milliseconds, start_event, stop_event);
    cout << "Conv13: " << milliseconds / 1000 << endl;
    totalTime += milliseconds / 1000;
    hipFree(Conv_Layer12_Features);

    //pool layer 13
    float * Pool_Layer13_Features ;
    hipMalloc(&Pool_Layer13_Features, 7 * 7 * 512 * sizeof(float));
    hipEventRecord(start_event, 0);
    pool_2<<<dim3(1, 1, 1), dim3(8, 8, 1)>>>(Conv_Layer13_Features, Pool_Layer13_Features, 14, 14 * 512);
    CHECK(hipDeviceSynchronize());
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&milliseconds, start_event, stop_event);
    cout << "Pool13: " << milliseconds / 1000 << endl;
    totalTime += milliseconds / 1000;
    hipFree(Conv_Layer13_Features);

    //fully connected layer 14
    float *Fu_Layer14_Features;
    float *Layer14_Weights_GPU, *Layer14_Weights_Bias_GPU;
    hipMalloc(&Layer14_Weights_GPU, LAYER14_PARAMS * sizeof(float));
    hipMalloc(&Layer14_Weights_Bias_GPU, LAYER14_BIAS_PARAMS * sizeof(float));
    hipMalloc(&Fu_Layer14_Features, 4096 * sizeof(float));

    hipMemcpy(Layer14_Weights_GPU, Layer14_Weights_CPU, LAYER14_PARAMS * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(Layer14_Weights_Bias_GPU, Layer14_Weights_Bias_CPU, LAYER14_BIAS_PARAMS * sizeof(float), hipMemcpyHostToDevice);

    hipEventRecord(start_event, 0);
    fn<<<128, 32>>>(Pool_Layer13_Features, Fu_Layer14_Features, Layer14_Weights_GPU, Layer14_Weights_Bias_GPU, 4096, 7 * 7 * 512);
    CHECK(hipDeviceSynchronize());
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&milliseconds, start_event, stop_event);
    cout << "FC14: " << milliseconds / 1000 << endl;
    totalTime += milliseconds / 1000;

    hipFree(Pool_Layer13_Features);
    hipFree(Layer14_Weights_GPU);
    hipFree(Layer14_Weights_Bias_GPU);

    //fully connected layer 15
    float *Fu_Layer15_Features;
    float *Layer15_Weights_GPU, *Layer15_Weights_Bias_GPU;
    hipMalloc(&Layer15_Weights_GPU, LAYER15_PARAMS * sizeof(float));
    hipMalloc(&Layer15_Weights_Bias_GPU, LAYER15_BIAS_PARAMS * sizeof(float));
    hipMalloc(&Fu_Layer15_Features, 4096 * sizeof(float));

    hipMemcpy(Layer15_Weights_GPU, Layer15_Weights_CPU, LAYER15_PARAMS * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(Layer15_Weights_Bias_GPU, Layer15_Weights_Bias_CPU, LAYER15_BIAS_PARAMS * sizeof(float), hipMemcpyHostToDevice);

    hipEventRecord(start_event, 0);
    fn<<<128, 32>>>(Fu_Layer14_Features, Fu_Layer15_Features, Layer15_Weights_GPU, Layer15_Weights_Bias_GPU, 4096, 4096);
    CHECK(hipDeviceSynchronize());
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&milliseconds, start_event, stop_event);
    cout << "FC15: " << milliseconds / 1000 << endl;
    totalTime += milliseconds / 1000;

    hipFree(Fu_Layer14_Features);
    hipFree(Layer15_Weights_GPU);
    hipFree(Layer15_Weights_Bias_GPU);

    //fully connected layer 16
    float * Fu_Layer16_Features;
    float * Layer16_Weights_GPU, *Layer16_Weights_Bias_GPU;
    hipMalloc(&Layer16_Weights_GPU, LAYER16_PARAMS * sizeof(float));
    hipMalloc(&Layer16_Weights_Bias_GPU, LAYER16_BIAS_PARAMS * sizeof(float));
    hipMalloc(&Fu_Layer16_Features, 1000 * sizeof(float));

    hipMemcpy(Layer16_Weights_GPU, Layer16_Weights_CPU, LAYER16_PARAMS * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(Layer16_Weights_Bias_GPU, Layer16_Weights_Bias_CPU, LAYER16_BIAS_PARAMS * sizeof(float), hipMemcpyHostToDevice);

    hipEventRecord(start_event, 0);
    fn<<<32, 32>>>(Fu_Layer15_Features, Fu_Layer16_Features, Layer16_Weights_GPU, Layer16_Weights_Bias_GPU, 1000, 4096);
    CHECK(hipDeviceSynchronize());
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&milliseconds, start_event, stop_event);
    cout << "FC16: " << milliseconds / 1000 << endl;
    totalTime += milliseconds / 1000;

    hipFree(Fu_Layer15_Features);
    hipFree(Layer16_Weights_GPU);
    hipFree(Layer16_Weights_Bias_GPU);

    //predict layer
    uint32_t * pred = (uint32_t*)malloc(sizeof(uint32_t));
    uint32_t * d_pred;
    hipMalloc(&d_pred, sizeof(uint32_t));
    hipEventRecord(start_event, 0);
    predict<<<1, 1024>>>(Fu_Layer16_Features, d_pred);
    CHECK(hipDeviceSynchronize());
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&milliseconds, start_event, stop_event);
    cout << "Pred: " << milliseconds / 1000 << endl;
    totalTime += milliseconds / 1000;
    cout << "Total Time: " << totalTime << endl;
    hipMemcpy(pred, d_pred, sizeof(uint32_t), hipMemcpyDeviceToHost);
    hipFree(Fu_Layer16_Features);
    cout << endl << "prediction : " << *pred << endl;

    CHECK(hipDeviceReset());
    return EXIT_SUCCESS;
}	
